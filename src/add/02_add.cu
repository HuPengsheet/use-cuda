#include<stdio.h>
#include<hip/hip_runtime.h>


/*
N=1000 ,gpu totoal time = 0.076800 ms
N=10000 ,gpu totoal time = 0.005120 ms
N=100000 ,gpu totoal time = 0.012608 ms
N=1000000 ,gpu totoal time = 0.087520 ms
N=10000000 ,gpu totoal time = 0.224096 ms
N=100000000 ,gpu totoal time = 1.675680 ms
*/

void init_array(float * array,float x,int n)
{
    for(int i=0;i<n;i++)
    {
        array[i] = x;
    }
}

void compare_array(const float *x,const float * y, int n)
{

    for(int i=0;i<n;i++)
    {
        if(x[i]-y[i]>0.01)
        {
            printf("faiulre \n");
            break;
        }
    }
}

__global__ void cuda_add(float * x, float * y, float *z,size_t n)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    for(int i = idx; i < n / 4; i += blockDim.x * gridDim.x)
    {   
        float4 a = reinterpret_cast<float4*>(x)[idx];
        float4 b = reinterpret_cast<float4*>(y)[idx];
        float4 c;
        c.x = a.x+b.x;
        c.y = a.y+b.y;
        c.z = a.z+b.z;
        c.w = a.x+b.w;
        reinterpret_cast<float4*>(z)[idx] = c;
    }

}

void test_add()
{
    for(int N=1000;N<=100000000;N=N*10)
    {
        float *hx,*hy,*hz,*dx,*dy,*dz;
        float *res;
        size_t nbytes = N*sizeof(float);
    
        hx = (float*)malloc(nbytes);
        hy = (float*)malloc(nbytes);
        hz = (float*)malloc(nbytes);
        init_array(hx,1.2,N);
        init_array(hy,1.8,N);
        init_array(hz,0,N);
    
    
    
    
        hipMalloc(&dx,nbytes);
        hipMalloc(&dy,nbytes);
        hipMalloc(&dz,nbytes);
    
        hipMemcpy(dx,hx,nbytes,hipMemcpyHostToDevice);
        hipMemcpy(dy,hy,nbytes,hipMemcpyHostToDevice);
    
        hipEvent_t begin, stop;
        hipEventCreate(&begin);
        hipEventCreate(&stop);
        
        float milliseconds=0;
        const int block_size = 512;
        const int grid_size = (N + block_size -1)/ block_size;
        hipEventRecord(begin);
        cuda_add<<<grid_size,block_size>>>(dx,dy,dz,N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, begin, stop);
        printf("N=%d ,gpu totoal time = %f ms\n",N,milliseconds);
    
        res = (float*)malloc(nbytes);
        hipMemcpy(res,dz,nbytes,hipMemcpyDeviceToHost);
    
    
        compare_array(hz,res,N);
    
        free(hx);
        free(hy);
        free(hz);
        hipFree(dx);
        hipFree(dy);
        hipFree(dz);
        hipDeviceSynchronize();
    }
}


int main()
{
    test_add();
    return 0;   
}