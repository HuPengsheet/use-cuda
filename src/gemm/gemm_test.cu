#include<stdio.h>
#include<random>
#include<hip/hip_runtime.h>






void compare_array(float * x,float *y,int n)
{
    int i=0;
    for(;i<n;i++)
    {
        if(abs(x[i]-y[i])>1e-3)
         {
            //printf("x[%d] = %f,   y[%d] = %f  \n",i,x[i],i,y[i]);
            printf("x and y not equal !\n");
            break;
         }
    }

    if(i==n)  printf("x = y \n");
}

//input a m*k
//input b k*n
//output c m*n
void sgemm(size_t m,size_t n,size_t k,float* a,float* b,float* c,float* bias)
{
    for(int i=0;i<m;i++)
    {
        float bia = bias[i];   
        for(int j=0;j<n;j++)
        {
            float sum = 0.0;
            for(int z=0;z<k;z++)
            {
                sum += a[i*k+z]*b[z*n+j];
            }

            c[i*n+j] = sum+bia;
        }
    }
}



// template <int BLOCK>
// __global__ void cuda_sgemm_forward(size_t m,size_t n,size_t k,float* a,float* b,float* c,float* bias)
// {
//     const int tx = threadIdx.x;
//     const int ty = threadIdx.y;
//     const int bx = blockIdx.x;
//     const int by = blockIdx.y;
  
//     float *begin_a = a + by * BLOCK * k;
//     float *begin_b = b + bx * BLOCK;
//     float *end_a = begin_a + k;
  
//     float sum = 0.f;
//     for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
//          a_ptr += BLOCK, b_ptr += BLOCK * n) {
//       __shared__ float ashare[BLOCK][BLOCK];
//       __shared__ float bshare[BLOCK][BLOCK];
  
//       ashare[ty][tx] = a_ptr[ty * k + tx];
//       bshare[ty][tx] = b_ptr[ty * n + tx];
//       __syncthreads();
  
//   #pragma unroll
//       for (int kk = 0; kk < BLOCK; ++kk) {
//         sum += ashare[ty][kk] * bshare[kk][tx];
//       }
//       __syncthreads();
//     }
  
//     c[(BLOCK * by + ty) * n + BLOCK * bx + tx] = sum;
// }

template <int BLOCK>
__global__ void cuda_sgemm_forward(size_t m,size_t n,size_t k,float* a,float* b,float* c,float *bias)
{

    

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    //printf("bx=%d by=%d \n",bx,by);
    int global_idx= bx*BLOCK+tx;
    int global_idy= by*BLOCK+ty;

    float *begin_a = a + bx * BLOCK * k;
    float *end_a = begin_a + k;
    float *a_bottom = a+(m-1)*k;
    float *a_block_bottom = begin_a+(BLOCK-1)*k;
    int a_x_gap = (a_bottom>=a_block_bottom) ? BLOCK : (BLOCK-(a_block_bottom-a_bottom)/k);


    float *begin_b = b + by * BLOCK;
    float *end_b = b+(k-1)*n;
    float *b_right = b+n;
    float *b_block_right = begin_b+BLOCK;
    int b_y_gap = (b_right>=b_block_right) ? BLOCK : (BLOCK-(b_block_right-b_right));
        

    float sum = 0.f;
    bool flag=1;
    for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
         a_ptr += BLOCK, b_ptr += BLOCK * n) {

        __shared__ float ashare[BLOCK][BLOCK];
        __shared__ float bshare[BLOCK][BLOCK];
        __shared__ float bias_share[BLOCK];
    
        float* a_block_right = a_ptr+BLOCK;
        int a_y_gap = (end_a>=a_block_right) ? BLOCK : (BLOCK-(a_block_right-end_a));

        float* b_block_bottom = b_ptr+(BLOCK-1) * n;
        int b_x_gap = (end_b>=b_block_bottom) ? BLOCK : (BLOCK-(b_block_bottom-end_b)/n);

    
        

        if(tx<a_x_gap&&ty<a_y_gap) ashare[tx][ty] = a_ptr[tx * k + ty];
        if(tx<b_x_gap&&ty<b_y_gap) bshare[tx][ty] = b_ptr[tx * n + ty];
        if(tx<a_x_gap) bias_share[tx] = bias[(begin_a-a)/k+tx];
        __syncthreads();

        // if(num==1&&global_idx==20&&global_idy==5)
        // {
        //     for(int i=0;i<BLOCK;i++)
        //     {
        //         for(int j=0;j<BLOCK;j++)
        //         {
        //             printf("%1.f ",ashare[i][j]);
        //         }
        //         printf("\n");
        //     } 
        // }

        // if(num==1&&global_idx==20&&global_idy==5) printf("**************\n");

        // //if(num==1&&global_idx==20&&global_idy==5) printf("a_x=%d a_y=%d b_x=%d by=%d\n",a_x_gap,a_y_gap,b_x_gap,b_y_gap);

        // if(num==1&&global_idx==20&&global_idy==5)
        // {
        //     for(int i=0;i<BLOCK;i++)
        //     {
        //         for(int j=0;j<BLOCK;j++)
        //         {
        //             printf("%1.f ",bshare[i][j]);
        //         }
        //         printf("\n");
        //     } 
        // }  

        // if(num==1&&global_idx==0&&global_idy==0)
        // {
        //     for(int i=0;i<BLOCK;i++)
        //     {
        //         for(int j=0;j<BLOCK;j++)
        //         {
        //             printf("%1.f ",ashare[i][j]);
        //         }
        //         printf("\n");
        //     } 
        // }

        // if(num==1&&global_idx==16&&global_idy==16) printf("**************\n");



  #pragma unroll
      for (int kk = 0; kk < BLOCK; ++kk) {
        sum += ashare[tx][kk] * bshare[kk][ty];
      }
      if(flag) 
      {
          sum+=bias_share[tx];
          flag=0;
      }
      __syncthreads();

      ashare[tx][ty]=0;
      bshare[tx][ty]=0; 
      __syncthreads();
    }

    if(global_idx<m&&global_idy<n)
    {
        // printf("global_idx=%d global_idy=%d sum=%f\n",global_idx,global_idy,sum);
        c[global_idx*n+global_idy] = sum;
    }

}
void cuda_sgemm(size_t m,size_t n,size_t k,float* h_a,float* h_b,float* h_c,float* h_bias)
{
    float *d_a,*d_b,*d_c,*d_bias;
    size_t a_nbytes = m*k*sizeof(float);
    size_t b_nbytes = n*k*sizeof(float);
    size_t c_nbytes = m*n*sizeof(float);
    size_t d_nbytes = m*sizeof(float);

    hipMalloc(&d_a,a_nbytes);
    hipMalloc(&d_b,b_nbytes);
    hipMalloc(&d_c,c_nbytes);
    hipMalloc(&d_bias,d_nbytes);

    hipMemcpy(d_a,h_a,a_nbytes,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,b_nbytes,hipMemcpyHostToDevice);
    hipMemcpy(d_bias,h_bias,d_nbytes,hipMemcpyHostToDevice);

    hipEvent_t begin, stop;
    hipEventCreate(&begin);
    hipEventCreate(&stop);
    
    float milliseconds=0;

    constexpr int BLOCK = 16;
    // subm, subn, subk
    dim3 block(BLOCK, BLOCK);
    dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);
  
    
    hipEventRecord(begin);
    cuda_sgemm_forward<BLOCK><<<grid,block>>>(m,n,k,d_a,d_b,d_c,d_bias);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, begin, stop);
    printf("m=%ld,n=%ld,k=%ld,  gpu totoal time = %f ms\n",m,n,k,milliseconds);


    hipMemcpy(h_c,d_c,c_nbytes,hipMemcpyDeviceToHost);
    // for (int i = 0; i < m; i++)
    // {
    //     for (int j = 0; j < n; j++)
    //     {
    //         printf("%1.f ",h_c[i * n + j]);
    //     }
    //     printf("\n");
    // }
    // /printf("******************gpu result\n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_bias);
}

//矩阵大小从 2 慢慢变大
void test_result()
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(1, 10);
    
    for (int m = 1, n = 3, k = 2; k <=4096; m = m + 10, n = n * 2, k = k * 2)
    {
        float *h_a, *h_b, *h_c,*h_bias, *result;
        size_t a_nbytes = m * k * sizeof(float);
        size_t b_nbytes = n * k * sizeof(float);
        size_t c_nbytes = n * m * sizeof(float);
        size_t d_nbytes = m * sizeof(float);
        

        h_a = (float *)malloc(a_nbytes);
        h_b = (float *)malloc(b_nbytes);
        h_c = (float *)malloc(c_nbytes);
        h_bias = (float *)malloc(d_nbytes);
        result = (float *)malloc(c_nbytes);



        for (int i = 0; i < m; i++)
        {
            for (int j = 0; j < k; j++)
            {
                h_a[i * k + j] = dis(gen);
            }
        }

        // for (int i = 0; i < m; i++)
        // {
        //     for (int j = 0; j < k; j++)
        //     {
        //         printf("%1.f ",h_a[i * k + j]);
        //     }
        //     printf("\n");
        // }
        // printf("******************8\n");

        for (int i = 0; i < k; i++)
        {
            for (int j = 0; j < n; j++)
            {
                h_b[i * n + j] = dis(gen);
            }
        }

        // for (int i = 0; i < k; i++)
        // {
        //     for (int j = 0; j < n; j++)
        //     {
        //         printf("%1.f ",h_b[i * n + j]);
        //     }
        //     printf("\n");
        // }
        // printf("******************8\n");

        for (int i = 0; i < m; i++)
        {
            h_bias[i] =  dis(gen);
            //h_bias[i] =  0;
        }

        cuda_sgemm(m,n,k,h_a,h_b,h_c,h_bias);
        sgemm(m,n,k,h_a,h_b,result,h_bias);
        compare_array(result,h_c,m*n);

        // for (int i = 0; i < m; i++)
        // {
        //     for (int j = 0; j < n; j++)
        //     {
        //         printf("%1.f ",result[i * n + j]);
        //     }
        //     printf("\n");
        // }
        // printf("******************cpu result\n");

        free(h_a); 
        free(h_b); 
        free(h_bias);
        free(result);
    }
}

int main()
{


    test_result();
    //test_time();
    printf("01_gemm_naive  run  !!!\n");
    return 0;
}