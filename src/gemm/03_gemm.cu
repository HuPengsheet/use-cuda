#include<stdio.h>
#include<random>
#include<hip/hip_runtime.h>






void compare_array(float * x,float *y,int n)
{
    int i=0;
    for(;i<n;i++)
    {
        if(abs(x[i]-y[i])>1e-3)
         {
            //printf("x[%d] = %f,   y[%d] = %f  \n",i,x[i],i,y[i]);
            printf("x and y not equal !\n");
            break;
         }
    }

    //if(i==n)  printf("x = y \n");
}

//input a m*k
//input b k*n
//output c m*n
void sgemm(size_t m,size_t n,size_t k,float* a,float* b,float* c)
{
    for(int i=0;i<m;i++)
    {
        
        for(int j=0;j<n;j++)
        {
            float sum = 0.0;
            for(int z=0;z<k;z++)
            {
                sum += a[i*k+z]*b[z*n+j];
                //printf("%f  %f %d  %f \n",a[i*k+z],b[z*n+j],z*n+j,sum);
            }

            c[i*n+j] = sum;
        }
    }
}



template <int BLOCK,int STRIDE>
__global__ void cuda_sgemm_forward(size_t m,size_t n,size_t k,float* a,float* b,float* c)
{
  // blockIdx control subpanel matrix
  constexpr int STEP = BLOCK * STRIDE;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int bx = blockIdx.x;
  const int by = blockIdx.y;

  float *begin_a = a + by * STEP * k;
  float *begin_b = b + bx * STEP;
  float *end_a = begin_a + k;

  float sum[STRIDE][STRIDE] = {0.f};
  for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += STEP, b_ptr += STEP * n) {
    __shared__ float ashare[STEP][STEP];
    __shared__ float bshare[STEP][STEP];

    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        ashare[ty * STRIDE + i][tx * STRIDE + j] =
            a_ptr[(ty * STRIDE + i) * k + tx * STRIDE + j];
        bshare[ty * STRIDE + i][tx * STRIDE + j] =
            b_ptr[(ty * STRIDE + i) * n + tx * STRIDE + j];
      }
    }
    __syncthreads();

#pragma unroll
    for (int i = 0; i < STRIDE; ++i) {
      for (int j = 0; j < STRIDE; ++j) {
        for (int kk = 0; kk < STEP; ++kk) {
          sum[i][j] +=
              ashare[ty * STRIDE + i][kk] * bshare[kk][tx * STRIDE + j];
        }
      }
    }

    __syncthreads();
  }

  for (int i = 0; i < STRIDE; ++i) {
    for (int j = 0; j < STRIDE; ++j) {
      c[(STEP * by + ty * STRIDE + i) * n + STEP * bx + tx * STRIDE + j] =
          sum[i][j];
    }
  }
}

void cuda_sgemm(size_t m,size_t n,size_t k,float* h_a,float* h_b,float* h_c)
{
    float *d_a,*d_b,*d_c;
    size_t a_nbytes = m*k*sizeof(float);
    size_t b_nbytes = n*k*sizeof(float);
    size_t c_nbytes = m*n*sizeof(float);


    hipMalloc(&d_a,a_nbytes);
    hipMalloc(&d_b,b_nbytes);
    hipMalloc(&d_c,c_nbytes);

    hipMemcpy(d_a,h_a,a_nbytes,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,b_nbytes,hipMemcpyHostToDevice);

    hipEvent_t begin, stop;
    hipEventCreate(&begin);
    hipEventCreate(&stop);
    
    float milliseconds=0;

    constexpr int BLOCK = 16;
    constexpr int STRIDE = 2;
    // subm, subn, subk
    dim3 block(BLOCK, BLOCK);
    dim3 grid((m + BLOCK - 1) / BLOCK/STRIDE,(n + BLOCK - 1) / BLOCK/STRIDE);
  
   
    hipEventRecord(begin);
    cuda_sgemm_forward<BLOCK,STRIDE><<<grid,block>>>(m,n,k,d_a,d_b,d_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, begin, stop);
    
    printf("m=%ld,n=%ld,k=%ld,  gpu totoal time = %f ms\n",m,k,n,milliseconds);


    hipMemcpy(h_c,d_c,c_nbytes,hipMemcpyDeviceToHost);


    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

//�����С�� 128 256 512 1024 2048 4096 
void test_time()
{
    std::random_device rd;  
    std::mt19937 gen(rd());  
    std::uniform_int_distribution<> dis(1, 10);  

    

    for(int i=128;i<=16384;i=i*2)
    {   
        //printf("start i= %d  \n",i);
        size_t nbytes = i*i*sizeof(float);
        float * h_a,*h_b,*h_c;

        h_a = (float *)malloc(nbytes);
        h_b = (float *)malloc(nbytes);
        h_c = (float *)malloc(nbytes);
        
        float *result = (float *)malloc(nbytes);
        //printf("mid i= %d  \n",i);
        for(int j=0;j<i*i;j++)
        {
            h_a[j] = dis(gen);
            h_b[j] = dis(gen);  
        }

        cuda_sgemm(i,i,i,h_a,h_b,h_c);
        // sgemm(i,i,i,h_a,h_b,result);
        // compare_array(result,h_c,i*i);
        

        free(h_a);
        free(h_b);
        free(h_c);
    }
}

int main()
{
    // float * out = (float *)malloc(12*sizeof(float));
    // //sgemm(3,4,3,a,b,out);
    
    // cuda_sgemm(3,4,3,a,b,out);
    //compare_array(out,c,12);

    test_time();
    test_time();
    printf("01_gemm_naive  run  !!!\n");
    return 0;
}