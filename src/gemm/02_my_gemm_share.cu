#include<stdio.h>
#include<random>
#include<hip/hip_runtime.h>






void compare_array(float * x,float *y,int n)
{
    int i=0;
    for(;i<n;i++)
    {
        if(abs(x[i]-y[i])>1e-3)
         {
            //printf("x[%d] = %f,   y[%d] = %f  \n",i,x[i],i,y[i]);
            printf("x and y not equal !\n");
            break;
         }
    }

    //if(i==n)  printf("x = y \n");
}

//input a m*k
//input b k*n
//output c m*n
void sgemm(size_t m,size_t n,size_t k,float* a,float* b,float* c,float* bias)
{
    for(int i=0;i<m;i++)
    {
        float bia = bias[i];   
        for(int j=0;j<n;j++)
        {
            float sum = 0.0;
            for(int z=0;z<k;z++)
            {
                sum += a[i*k+z]*b[z*n+j];
            }

            c[i*n+j] = sum+bia;
        }
    }
}



// template <int BLOCK>
// __global__ void cuda_sgemm_forward(size_t m,size_t n,size_t k,float* a,float* b,float* c,float* bias)
// {
//     int _m = blockIdx.x * BLOCK + threadIdx.x;
//     int _n = blockIdx.y * BLOCK + threadIdx.y;
//     if (_m < m and _n < n) {
//       float sum = 0.f;
//       for (int i = 0; i < k; ++i) {
//         sum += a[_m * k + i] * b[i * n + _n];
//       }
//       c[_m * n + _n] = sum+bias[_m];
//     }
// }


template <int BLOCK>
__global__ void cuda_sgemm_forward(size_t m,size_t n,size_t k,float* a,float* b,float* c,float *bias)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    int global_idx= bx*BLOCK+tx;
    int global_idy= by*BLOCK+ty;

    float *begin_a = a + by * BLOCK * k;
    float *begin_b = b + bx * BLOCK;
    float *end_a = begin_a + k;
    
    int num=0;
    float sum = 0.f;
    for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
         a_ptr += BLOCK, b_ptr += BLOCK * n) {
            num++;
      __shared__ float ashare[BLOCK][BLOCK];
      __shared__ float bshare[BLOCK][BLOCK];
      //printf("global_idy=%d ,global_idx=%d  \n",global_idy,global_idx);
    //   printf("num= %d \n",num);
      if(num==2)
      {
        if(global_idy<m&&global_idx<n){
            ashare[ty][tx] = a_ptr[ty * k + tx];
            bshare[ty][tx] = b_ptr[ty * n + tx];
            printf("global_idy=%d ,global_idx=%d  \n",global_idy,global_idx);
        }
      }
      __syncthreads();

    //   if(num==1)
    //   {
    //     if(global_idx==0&&global_idy==0){
    //         for(int i=0;i<BLOCK;i++){
    //             for(int j=0;j<BLOCK;j++)
    //             {
    //                 printf("%f ",ashare[i][j]);
    //             }
    //             printf("\n");
    //         }
    //       }
    //   }


    //   if(global_idx==1&&global_idy==1) printf("**************\n");

    //   if(global_idx==1&&global_idy==1){
    //     for(int i=0;i<BLOCK;i++){
    //         for(int j=0;j<BLOCK;j++)
    //         {
    //             printf("%f ",bshare[i][j]);
    //         }
    //         printf("\n");
    //     }
    //   }
    //   if(global_idx==1&&global_idy==1) printf("**************\n");
     

  #pragma unroll
      for (int kk = 0; kk < BLOCK; ++kk) {
        sum += ashare[ty][kk] * bshare[kk][tx];
      }
      __syncthreads();
    }
    if(global_idy<m&&global_idx<n)
    {
        c[global_idy*m+global_idx] = sum+bias[global_idy];
    }
    
}

void cuda_sgemm(size_t m,size_t n,size_t k,float* h_a,float* h_b,float* h_c,float* h_bias)
{
    float *d_a,*d_b,*d_c,*d_bias;
    size_t a_nbytes = m*k*sizeof(float);
    size_t b_nbytes = n*k*sizeof(float);
    size_t c_nbytes = m*n*sizeof(float);
    size_t d_nbytes = m*sizeof(float);

    hipMalloc(&d_a,a_nbytes);
    hipMalloc(&d_b,b_nbytes);
    hipMalloc(&d_c,c_nbytes);
    hipMalloc(&d_bias,d_nbytes);

    hipMemcpy(d_a,h_a,a_nbytes,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,b_nbytes,hipMemcpyHostToDevice);
    hipMemcpy(d_bias,h_bias,d_nbytes,hipMemcpyHostToDevice);

    hipEvent_t begin, stop;
    hipEventCreate(&begin);
    hipEventCreate(&stop);
    
    float milliseconds=0;

    constexpr int BLOCK = 16;
    // subm, subn, subk
    dim3 block(BLOCK, BLOCK);
    dim3 grid((m + BLOCK - 1) / BLOCK, (n + BLOCK - 1) / BLOCK);
  
    
    hipEventRecord(begin);
    cuda_sgemm_forward<BLOCK><<<grid,block>>>(m,n,k,d_a,d_b,d_c,d_bias);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, begin, stop);
    printf("m=%ld,n=%ld,k=%ld,  gpu totoal time = %f ms\n",m,k,n,milliseconds);


    hipMemcpy(h_c,d_c,c_nbytes,hipMemcpyDeviceToHost);

    // for(int i=0;i<m;i++){
    //     for(int j=0;j<n;j++)
    //     {
    //         printf("%f ",h_c[i*m+j]);
    //     }
    //     printf("\n");
    // }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_bias);
}

//矩阵大小从 2 慢慢变大
void test_time()
{
    std::random_device rd;  
    std::mt19937 gen(rd());  
    std::uniform_int_distribution<> dis(1, 10);  

    

    for(int i=17;i<=17;i=i+1)
    {   
        //printf("start i= %d  \n",i);
        size_t nbytes = i*i*sizeof(float);
        float * h_a,*h_b,*h_c,*h_bias;

        h_a = (float *)malloc(nbytes);
        h_b = (float *)malloc(nbytes);
        h_c = (float *)malloc(nbytes);
        h_bias = (float *)malloc(nbytes);
        
        float *result = (float *)malloc(nbytes);
        //printf("mid i= %d  \n",i);
        for(int j=0;j<i*i;j++)
        {
            h_a[j] = dis(gen);
            //printf("a[%d]=%f ",j,h_a[j]);
            //printf("\n");
            h_b[j] = dis(gen);  
        }
        
        for(int k=0;k<i;k++)
        {
            h_bias[k]=0;
        }

        cuda_sgemm(i,i,i,h_a,h_b,h_c,h_bias);
        
        if(i<=4096) 
        {
            sgemm(i,i,i,h_a,h_b,result,h_bias);
            compare_array(result,h_c,i*i);
        }

        free(h_a);
        free(h_b);
        free(h_c);
    }
}

int main()
{


    test_time();
    //test_time();
    printf("01_gemm_naive  run  !!!\n");
    return 0;
}